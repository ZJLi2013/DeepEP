#include "hip/hip_runtime.h"
#include <vector>
#include <cstring>

#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "utils.cuh"

#ifndef DISABLE_NVSHMEM
#include "ibgda_device.cuh"
#endif

namespace deep_ep {

namespace intranode {

template<int kNumRanks>
__global__ void barrier(int** barrier_signal_ptrs, int rank) {
    barrier_block<kNumRanks>(barrier_signal_ptrs, rank);
}

void barrier(int** barrier_signal_ptrs, int rank, int num_ranks, hipStream_t stream) {
#define BARRIER_LAUNCH_CASE(ranks) \
    LAUNCH_KERNEL(&cfg, barrier<ranks>, barrier_signal_ptrs, rank); \
    break

    SETUP_LAUNCH_CONFIG(1, 32, stream);
    SWITCH_RANKS(BARRIER_LAUNCH_CASE);
#undef BARRIER_LAUNCH_CASE
}

} // namespace intranode

namespace internode {

#ifndef DISABLE_NVSHMEM
nvshmem_team_t cpu_rdma_team = NVSHMEM_TEAM_INVALID;
nvshmem_team_config_t cpu_rdma_team_config;

std::vector<uint8_t> get_unique_id() {
    nvshmemx_uniqueid_t unique_id;
    nvshmemx_get_uniqueid(&unique_id);
    std::vector<uint8_t> result(sizeof(nvshmemx_uniqueid_t));
    std::memcpy(result.data(), &unique_id, sizeof(nvshmemx_uniqueid_t));
    return result;
}

int init(const std::vector<uint8_t> &root_unique_id_val, int rank, int num_ranks, bool low_latency_mode) {
    nvshmemx_uniqueid_t root_unique_id;
    nvshmemx_init_attr_t attr;
    std::memcpy(&root_unique_id, root_unique_id_val.data(), sizeof(nvshmemx_uniqueid_t));
    nvshmemx_set_attr_uniqueid_args(rank, num_ranks, &root_unique_id, &attr);
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr);

    /*
        通过 root_unique_id 来构建nvshmem的通信组
        root_unique_id 是 rdma_rank=0 的 8张显卡会各自产生 unique_id, 
        然后每台机器上的显卡使用 rdma_group = rank % 8 来获取 rdma_rank=0的8个显卡对应的第rdma_group显卡的unique id
        也就是说，rdma_rank 分组不是同一个node中的8张gpu，而是 所有 nodes上，相同 local_gpu_id 的分组
        然后通过nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr)完成网络组构建
    */

    // Create sub-RDMA teams
    // NOTES: if `num_ranks <= NUM_MAX_NVL_PEERS` then only low-latency kernels are used
    if (low_latency_mode and num_ranks > NUM_MAX_NVL_PEERS) {
        EP_HOST_ASSERT(cpu_rdma_team == NVSHMEM_TEAM_INVALID);
        EP_HOST_ASSERT(num_ranks % NUM_MAX_NVL_PEERS == 0);
        EP_HOST_ASSERT(nvshmem_team_split_strided(NVSHMEM_TEAM_WORLD, rank % NUM_MAX_NVL_PEERS, NUM_MAX_NVL_PEERS,
                                                  num_ranks / NUM_MAX_NVL_PEERS, &cpu_rdma_team_config, 0, &cpu_rdma_team) == 0);
        EP_HOST_ASSERT(cpu_rdma_team != NVSHMEM_TEAM_INVALID);
    }

    nvshmem_barrier_all();
    return nvshmem_my_pe();
}

void* alloc(size_t size, size_t alignment) {
    return nvshmem_align(alignment, size);
}
/*
    aligned allocations for inter-node communication buffers, used for : 
        * RDMA operations in the `internode` namepsace
        * buffer for gpu-to-gpu communication across nodes
        * preparing memory used fornvshmem Put/Get operations and atomics
    * must be freed with `nvshmem_free()`
*/

void free(void* ptr) {
    nvshmem_free(ptr);
}

void barrier() {
    nvshmem_barrier_all();
}

void finalize() {
    if (cpu_rdma_team != NVSHMEM_TEAM_INVALID) {
        nvshmem_team_destroy(cpu_rdma_team);
        cpu_rdma_team = NVSHMEM_TEAM_INVALID;
    }
    nvshmem_finalize();
}
#endif

} // namespace internode

} // namespace deep_ep
